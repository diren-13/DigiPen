#include "hip/hip_runtime.h"
/* Start Header *****************************************************************/ 

/*! \file   kernel.cu

    \author Diren D Bharwani, diren.dbharwani, 390002520 

    \par    diren.dbharwani@digipen.edu

    \date   Oct 2, 2022 

    \brief  Copyright (C) 2022 DigiPen Institute of Technology.
            Reproduction or disclosure of this file or its contents without the
            prior written consent of DigiPen Institute of Technology is prohibited.

FUCK CUDA
*/ 

/* End Header *******************************************************************/

#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "helper.h"

//////////////////////////////////////////////////////////////////////////////////

//P and M column-major, N row-major
__global__ void matrixMultiply
(
  FLOAT_TYPE*       P   //<! [out]  an mxn matrix
, const FLOAT_TYPE* A   //<! [in]   an mxk matrix
, const FLOAT_TYPE* B   //<! [in]   an kxn matrix
, const int         m
, const int         n
, const int         k
) 
{
  // Shared memory for tiling input N array
  __shared__ FLOAT_TYPE B_s[TILE_WIDTH_RATIO_K][TILE_WIDTH_N];

  // Output array
  FLOAT_TYPE pReg[TILE_WIDTH_N] { 0.0f };

  for (int nIter = 0; nIter < (k - 1) / TILE_WIDTH_RATIO_K + 1; ++nIter)
  {
    // Load data into shared memory
    const unsigned int B_ROW = nIter * TILE_WIDTH_RATIO_K + threadIdx.x / TILE_WIDTH_N;
    const unsigned int B_COL = blockIdx.y * TILE_WIDTH_N + threadIdx.x % TILE_WIDTH_N;
    const unsigned int B_IDX = B_ROW * n + B_COL;

    const unsigned int SHARED_ROW = threadIdx.x / TILE_WIDTH_N;
    const unsigned int SHARED_COL = threadIdx.x % TILE_WIDTH_N;

    const bool WITHIN_B_BOUNDARY = static_cast<int>(B_ROW) < k && static_cast<int>(B_COL) < n;
    B_s[SHARED_ROW][SHARED_COL] = WITHIN_B_BOUNDARY ? B[B_IDX] : 0.0f;

    __syncthreads();

    // Calculate Partial Sums
    for (int i = 0; i < TILE_WIDTH_RATIO_K; ++i)
    {
      const unsigned int A_ROW = blockIdx.x * TILE_WIDTH_M + threadIdx.x;
      const unsigned int A_COL = i + nIter * TILE_WIDTH_RATIO_K;
      const unsigned int A_IDX = A_COL * m + A_ROW;

      const bool WITHIN_A_BOUNDARY = static_cast<int>(A_ROW) < m && static_cast<int>(A_COL) < k;
      const FLOAT_TYPE M_REG = WITHIN_A_BOUNDARY ? A[A_IDX] : 0.0f;

      for (int j = 0; j < TILE_WIDTH_N; ++j)
        pReg[j] += M_REG * B_s[i][j];
    }

    __syncthreads();
  }

  for (int i = 0; i < TILE_WIDTH_N; ++i)
  {
    const unsigned int OUTPUT_ROW = blockIdx.x * TILE_WIDTH_M + threadIdx.x;
    const unsigned int OUTPUT_COL = blockIdx.y * TILE_WIDTH_N + i;
    //const unsigned int OUTPUT_IDX = OUTPUT_ROW * n + OUTPUT_COL;
    const unsigned int OUTPUT_IDX = OUTPUT_COL * m + OUTPUT_ROW;

    const bool WITHIN_OUTPUT_BOUNDARY = static_cast<int>(OUTPUT_ROW) < m && static_cast<int>(OUTPUT_COL) < n;
    atomicAdd(&P[OUTPUT_IDX], WITHIN_OUTPUT_BOUNDARY ? pReg[i] : static_cast<FLOAT_TYPE>(0));
  }
}

void matrixMultiplyGPU(FLOAT_TYPE* C,
  FLOAT_TYPE* A,
  FLOAT_TYPE* B,
  int numARows,
  int numBColumns,
  int numAColumns)
{
  //@@ Initialize the grid and block dimensions here

  dim3 dimGrid((numARows - 1) / TILE_WIDTH_M + 1, (numBColumns - 1) / TILE_WIDTH_N + 1);
  dim3 dimBlock(TILE_WIDTH_M, 1);

  matrixMultiply << <dimGrid, dimBlock >> > (C,
    A,
    B,
    numARows,
    numBColumns,
    numAColumns);

  getLastCudaError("matrixMultiply failed\n");
  hipDeviceSynchronize();
}
