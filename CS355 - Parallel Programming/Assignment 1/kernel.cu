#include "hip/hip_runtime.h"
/* Start Header *****************************************************************/ 

/*! \file kernel.cu

    \author Diren D Bharwani, diren.dbharwani, 390002520 

    \par    diren.dbharwani@digipen.edu

    \date   Sept 13, 2022 

    \brief  Copyright (C) 2022 DigiPen Institute of Technology.
            Reproduction or disclosure of this file or its contents without the
            prior written consent of DigiPen Institute of Technology is prohibited.

*/ 

/* End Header *******************************************************************/

#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 32
typedef unsigned int uint;

__global__
void heatDistrCalc(float* in, float* out, uint nRowPoints)
{
  const uint ROW = blockDim.y * blockIdx.y + threadIdx.y;
  const uint COL = blockDim.x * blockIdx.x + threadIdx.x;

  if (ROW < nRowPoints && COL < nRowPoints)
  {
    const uint IDX = ROW * nRowPoints + COL;

    const bool IS_EDGE = (ROW == 0) || (COL == 0) || (ROW == nRowPoints - 1) || (COL == nRowPoints - 1);
    if (IS_EDGE)
    {
      out[IDX] = in[IDX];
    }
    else
    {
      const uint T_IDX = (ROW + 1) * nRowPoints + COL;
      const uint B_IDX = (ROW - 1) * nRowPoints + COL;
      const uint R_IDX = ROW * nRowPoints + (COL + 1);
      const uint L_IDX = ROW * nRowPoints + (COL - 1);

      const float SUM = in[T_IDX] + in[B_IDX] + in[R_IDX] + in[L_IDX];
      out[ROW * nRowPoints + COL] = SUM * 0.25f;
    }
  }
}

///not required in A1
///Shared memory kernel function for heat distribution calculation
__global__
void heatDistrCalcShm(float* in, float* out, uint nRowPoints)
{

}

__global__
void heatDistrUpdate(float* in, float* out, uint nRowPoints)
{
  const uint ROW = blockDim.y * blockIdx.y + threadIdx.y;
  const uint COL = blockDim.x * blockIdx.x + threadIdx.x;

  if (ROW < nRowPoints && COL < nRowPoints)
  {
    const uint IDX = ROW * nRowPoints + COL;
    out[IDX] = in[IDX];
  }
}

extern "C" void heatDistrGPU
(
  float*  d_DataIn,
  float*  d_DataOut,
  uint    nRowPoints,
  uint    nIter
)
{
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 DimGrid2(ceil(((float)nRowPoints) / BLOCK_SIZE), ceil(((float)nRowPoints) / BLOCK_SIZE), 1);

  for (uint k = 0; k < nIter; k++) 
  {
    heatDistrCalc<<<DimGrid2, DimBlock>>>((float*)d_DataIn, (float*)d_DataOut, nRowPoints);
    getLastCudaError("heatDistrCalc failed\n");
    hipDeviceSynchronize();

    heatDistrUpdate<<<DimGrid2, DimBlock>>>((float*)d_DataOut, (float*)d_DataIn, nRowPoints);
    getLastCudaError("heatDistrUpdate failed\n");
    hipDeviceSynchronize();
  }
}
